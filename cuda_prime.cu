#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Generates a random unsigned long greater than `lower`
// `uradom` is a file descriptor pointing to `/dev/urandom`
unsigned long random_number(unsigned long lower, FILE* urandom) {
	unsigned long n = 0;
	while (n < lower) {
		size_t read = fread(&n, sizeof(unsigned long), 1, urandom);
		if (read != 1) {
			printf("Error reading `/dev/urandom`\n");
		}
	}
	if (n % 2 == 0) {
		n++;
	}
	return n;
}

// Tests if `n` is prime and stores the result in `p[threadIdx.x]`
__global__ void is_prime(const unsigned long n, bool p[]) {
	int  my_idx = threadIdx.x;
	unsigned long my_n = n + my_idx * 2;

	if (my_n <= 3) {
		p[my_idx] = my_n < 1;
		return;
	}
	if (my_n % 2 == 0) {
	printf("%lu is even\n", my_n);
		p[my_idx] = false;
		return;
	}
	if (my_n % 3 == 0) {
		p[my_idx] = false;
		return;
	}
	for (unsigned long i = 5; i * i < my_n; i += 6) {
		if (my_n % i == 0) {
			p[my_idx] = false;
			return;
		}
		if (my_n % (i + 2) == 0) {
			p[my_idx] = false;
			return;
		}
	}
	p[my_idx] = true;
	return;
}

int main() {

	FILE* urandom = fopen("/dev/urandom", "rb");
	if (urandom == NULL) {
		printf("Cannot open `/dev/urandom`\n");
		exit(1);
	}
	
	int blk_ct = 1;
	int th_per_blk = 32;
	unsigned long rand = random_number((unsigned long)1<<32, urandom);
	bool* is_p;
	bool noprime = true;

	hipMallocManaged(&is_p, th_per_blk*sizeof(bool));

	// Time varaibles
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	// Start timer here
	hipEventRecord(start);

	while (noprime) {
		is_prime<<<blk_ct, th_per_blk>>>(rand, is_p);
		hipDeviceSynchronize();

		for(int i=0; i<th_per_blk; i++) {
			if (is_p[i]) {
				printf("%d: %lu\n", is_p[i], rand+i*2);
				noprime = false;
				break;
			}
		}
	}

	// Stop timer
	hipEventRecord(stop);

	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	// Show time
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time elapsed: %f milliseconds\n", milliseconds);

	fclose(urandom);
	hipFree(is_p);
	return 0;
}
