#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>

/**
 * Generates a random unsigned long greater than `lower`
 * 
 * @param lower is lower bound of generated number
 * @param uradom is a file descriptor pointing to `/dev/urandom`
 */
unsigned long random_number(unsigned long lower, FILE* urandom) {
	unsigned long n = 0;
	while (n < lower) {
		size_t read = fread(&n, sizeof(unsigned long), 1, urandom);
		if (read != 1) {
			printf("Error reading `/dev/urandom`\n");
		}
	}
	if (n % 2 == 0) {
		n++;
	}
	return n;
}

/**
 * Tests if `n` is prime and stores the result in `p[threadIdx.x]`
 * 
 * @param n is unsigned long in that will be tested
 * @param p is a boolean array that stores t/f info
 * @param block_size is clearly the block size
 * @note This is cuda function
 */
__global__ void is_prime_part(const unsigned long n, bool p[], const unsigned long block_size) {
	int my_idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long lower = block_size * my_idx + 5;
	unsigned long upper = block_size * (my_idx + 1) + 5;
	p[my_idx] = true;
	for (unsigned long i = lower; i < upper; i += 6) {
		if (n % i == 0) {
			p[my_idx] = false;
			return;
		}
		if (n % (i + 2) == 0) {
			p[my_idx] = false;
			return;
		}
	}
}

/**
 * Tests if `n` is prime
 * 
 * @param n is unsigned long in that will be tested
 * @param blk_ct is the block number of cuda kernel
 * @param th_per_blk is the thread number of each block
 * @note This is also the function which calls __global__
 */
bool is_prime(const unsigned long n, int blk_ct, int th_per_blk) {

	if (n <= 3) {
		return n < 1;
	}
	if (n % 2 == 0) {
		return false;
	}
	if (n % 3 == 0) {
		return false;
	}

	unsigned long block_size = sqrt(n) / (blk_ct * th_per_blk);
	block_size += 6 - (block_size % 6);
	bool* p;
	hipMallocManaged(&p, blk_ct * th_per_blk * sizeof(bool));

	is_prime_part<<<blk_ct, th_per_blk>>>(n, p, block_size);

	hipDeviceSynchronize();
	for (int i = 0; i < blk_ct * th_per_blk; i++) {
		if (p[i] == false) {

			hipFree(p);
			return false;
		}
	}


	hipFree(p);
	return true;
}

/**
 * Get random number from urandom, initial cuda kernel,
 * and prints the result.
 * 
 * @note This also times the threads
 */
int main() {

	FILE* urandom = fopen("/dev/urandom", "rb");
	if (urandom == NULL) {
		printf("Cannot open `/dev/urandom`\n");
		exit(1);
	}
	
	//128 block * 512 thread per block = 65536 threads
	int blk_ct = 128;
	int th_per_blk = 512;
	unsigned long lower = (unsigned long)1 << 32 + 1;
	unsigned long rand = random_number(lower, urandom);

	// Time varaibles
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	// Start timer here
	hipEventRecord(start);

	while (true) {
		if (is_prime(rand, blk_ct, th_per_blk)) {
			printf("\nPrime num: %lu\n", rand);
			break;
		}
		rand += 2;
		if (rand < lower) {
			rand = lower;
		}
	}

	// Stop timer
	hipEventRecord(stop);
	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	// Show time
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time elapsed: %f milliseconds\n", milliseconds);

	fclose(urandom);
	return 0;
}
